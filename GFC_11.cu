#include "hip/hip_runtime.h"
/*
Copyright (c) 2011, Texas State University-San Marcos. All rights reserved.

Redistribution and use in source and binary forms, with or without modification,
are permitted for academic, research, experimental, or personal use provided
that the following conditions are met:

   * Redistributions of source code must retain the above copyright notice,
     this list of conditions and the following disclaimer.
   * Redistributions in binary form must reproduce the above copyright notice,
     this list of conditions and the following disclaimer in the documentation
     and/or other materials provided with the distribution.
   * Neither the name of Texas State University-San Marcos nor the names of its
     contributors may be used to endorse or promote products derived from this
     software without specific prior written permission.

For all other uses, please contact the Office for Commercialization and Industry
Relations at Texas State University-San Marcos <http://www.txstate.edu/ocir/>.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Authors: Molly A. O'Neil and Martin Burtscher
*/


#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <fstream>

#include "book.h"
#include "fileSeperatorMerger.h"

#define ull unsigned long long
#define MAX (64*1024*1024)

#define WARPSIZE 32
#define MAXWARP 32

__constant__ int dimensionalityd; // dimensionality parameter
__constant__ ull *cbufd; // ptr to uncompressed data
__constant__ unsigned char *dbufd; // ptr to compressed data
__constant__ ull *fbufd; // ptr to decompressed data
__constant__ int *cutd; // ptr to chunk boundaries
__constant__ int *offd; // ptr to chunk offsets after compression

/************************************************************************************/

/*
This is the GPU compression kernel, which should be launched using the block count
and warps/block:
  CompressionKernel<<<blocks, WARPSIZE*warpsperblock>>>();

Inputs
------
dimensionalityd: dimensionality of trace (from cmd line)
cbufd: ptr to the uncompressed data
cutd: ptr to array of chunk boundaries

Output
------
The compressed data, in dbufd 
Compressed chunk offsets for offset table, in offd
*/

/* adapted from Hacker's Delight */
__device__ int clzll(ull x)
{
  int n;
  if (x == 0) return(64);

  n = 0;
  if (x <= 0x00000000FFFFFFFFL) {n = n + 32; x = x << 32;}
  if (x <= 0x0000FFFFFFFFFFFFL) {n = n + 16; x = x << 16;}
  if (x <= 0x00FFFFFFFFFFFFFFL) {n = n + 8; x = x << 8;}
  if (x <= 0x0FFFFFFFFFFFFFFFL) {n = n + 4; x = x << 4;}
  if (x <= 0x3FFFFFFFFFFFFFFFL) {n = n + 2; x = x << 2;}
  if (x <= 0x7FFFFFFFFFFFFFFFL) {n = n + 1;}
  
  return n;
}

__global__ void CompressionKernel()
{
  register int offset, code, bcount, tmp, off, beg, end, lane, warp, iindex, lastidx, start, term;
  register ull diff, prev;
  __shared__ int ibufs[32 * (3 * WARPSIZE / 2)]; // shared space for prefix sum

  // index within this warp
  lane = threadIdx.x & 31;
  // index within shared prefix sum array
  iindex = threadIdx.x / WARPSIZE * (3 * WARPSIZE / 2) + lane;
  ibufs[iindex] = 0;
  iindex += WARPSIZE / 2;
  lastidx = (threadIdx.x / WARPSIZE + 1) * (3 * WARPSIZE / 2) - 1;
  // warp id
  warp = (threadIdx.x + blockIdx.x * blockDim.x) / WARPSIZE;
  // prediction index within previous subchunk
  offset = WARPSIZE - (dimensionalityd - lane % dimensionalityd) - lane;

  // determine start and end of chunk to compress
  start = 0;
  if (warp > 0) start = cutd[warp-1];
  term = cutd[warp];
  off = ((start+1)/2*17);

  prev = 0;
  for (int i = start + lane; i < term; i += WARPSIZE) {
    // calculate delta between value to compress and prediction
    // and negate if negative
    diff = cbufd[i] - prev;
    code = (diff >> 60) & 8;
    if (code != 0) {
      diff = -diff;
    }

    // count leading zeros in positive delta
    //bcount = 8 - (__clzll(diff) >> 3);
	/*
	* without __clzll
	*/
	bcount = 8 - (clzll(diff) >> 3);

    if (bcount == 2) bcount = 3; // encode 6 lead-zero bytes as 5

    // prefix sum to determine start positions of non-zero delta bytes
    ibufs[iindex] = bcount;
    __threadfence_block();
    ibufs[iindex] += ibufs[iindex-1];
    __threadfence_block();
    ibufs[iindex] += ibufs[iindex-2];
    __threadfence_block();
    ibufs[iindex] += ibufs[iindex-4];
    __threadfence_block();
    ibufs[iindex] += ibufs[iindex-8];
    __threadfence_block();
    ibufs[iindex] += ibufs[iindex-16];
    __threadfence_block();

    // write out non-zero bytes of delta to compressed buffer
    beg = off + (WARPSIZE/2) + ibufs[iindex-1];
    end = beg + bcount;
    for (; beg < end; beg++) {
      dbufd[beg] = diff;
      diff >>= 8;
    }

    if (bcount >= 3) bcount--; // adjust byte count for the dropped encoding
    tmp = ibufs[lastidx];
    code |= bcount;
    ibufs[iindex] = code;
    __threadfence_block();

    // write out half-bytes of sign and leading-zero-byte count (every other thread
    // writes its half-byte and neighbor's half-byte)
    if ((lane & 1) != 0) {
      dbufd[off + (lane >> 1)] = ibufs[iindex-1] | (code << 4);
    }
    off += tmp + (WARPSIZE/2);

    // save prediction value from this subchunk (based on provided dimensionality)
    // for use in next subchunk
    prev = cbufd[i + offset];
  }

  // save final value of off, which is total bytes of compressed output for this chunk
  if (lane == 31) offd[warp] = off;
}

/************************************************************************************/

/*
This is the GPU decompression kernel, which should be launched using the block count
and warps/block:
  CompressionKernel<<<blocks, WARPSIZE*warpsperblock>>>();

Inputs
------
dimensionalityd: dimensionality of trace
dbufd: ptr to array of compressed data
cutd: ptr to array of chunk boundaries

Output
------
The decompressed data in fbufd
*/

__global__ void DecompressionKernel()
{
  register int offset, code, bcount, off, beg, end, lane, warp, iindex, lastidx, start, term;
  register ull diff, prev;
  __shared__ int ibufs[32 * (3 * WARPSIZE / 2)];

  // index within this warp
  lane = threadIdx.x & 31;
  // index within shared prefix sum array
  iindex = threadIdx.x / WARPSIZE * (3 * WARPSIZE / 2) + lane;
  ibufs[iindex] = 0;
  iindex += WARPSIZE / 2;
  lastidx = (threadIdx.x / WARPSIZE + 1) * (3 * WARPSIZE / 2) - 1;
  // warp id
  warp = (threadIdx.x + blockIdx.x * blockDim.x) / WARPSIZE;
  // prediction index within previous subchunk
  offset = WARPSIZE - (dimensionalityd - lane % dimensionalityd) - lane;

  // determine start and end of chunk to decompress
  start = 0;
  if (warp > 0) start = cutd[warp-1];
  term = cutd[warp];
  off = ((start+1)/2*17);

  prev = 0;
  for (int i = start + lane; i < term; i += WARPSIZE) {

    // read in half-bytes of size and leading-zero count information
    if ((lane & 1) == 0) {
      code = dbufd[off + (lane >> 1)];
      ibufs[iindex] = code;
      ibufs[iindex + 1] = code >> 4;
    }
/*
/////////////////////////
//code = dbufd[off + (lane >> 1)]*((-1)*(lane&1)+1) + code*(lane&1);
//ibufs[iindex] = ((-1)*(lane&1)+1)*code + ibufs[iindex]*(lane&1);
    if ((lane & 1) == 0) {
//      code = dbufd[off + (lane >> 1)];
//      ibufs[iindex] = code;
      ibufs[iindex + 1] = code >> 4;
    }
*/	

//	ibufs[iindex + 1] = (code >> 4)*((-1)*(lane&1)+1) + ibufs[iindex + 1]*(lane&1);

    off += (WARPSIZE/2);
    __threadfence_block();
    code = ibufs[iindex];

    bcount = code & 7;
/////////////
    if (bcount >= 2) bcount++;
//bcount = ((bcount-2)>>31)*(-1)+1+bcount; 

    // calculate start positions of compressed data
    ibufs[iindex] = bcount;
    __threadfence_block();
    ibufs[iindex] += ibufs[iindex-1];
    __threadfence_block();
    ibufs[iindex] += ibufs[iindex-2];
    __threadfence_block();
    ibufs[iindex] += ibufs[iindex-4];
    __threadfence_block();
    ibufs[iindex] += ibufs[iindex-8];
    __threadfence_block();
    ibufs[iindex] += ibufs[iindex-16];
    __threadfence_block();

    // read in compressed data (the non-zero bytes)
    beg = off + ibufs[iindex-1];
    off += ibufs[lastidx];
    end = beg + bcount - 1;
    diff = 0;
    for (; beg <= end; end--) {
      diff <<= 8;
      diff |= dbufd[end];
    }

//we can remove this if by using this/////////////////////////
//(-0.25*(code&8)+1)*diff;

    // negate delta if sign bit indicates it was negated during compression
    if ((code & 8) != 0) {
      diff = -diff;
    }

//diff=(-0.25*(code&8)+1)*diff;

    // write out the uncompressed word
    fbufd[i] = prev + diff;
    __threadfence_block();

    // save prediction for next subchunk
    prev = fbufd[i + offset];
  }
}

/************************************************************************************/

static void CudaTest(char *msg)
{
  hipError_t e;

  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "%s: %d\n", msg, e);
    fprintf(stderr, "%s\n", hipGetErrorString(e));
    exit(-1);
  }
}

//this function is used to get the length of one file
int getFileLength(FILE* inputFile)
{
  int size=-1;

  if (inputFile==NULL)
  {
	std::cout<<"error open file"<<std::endl;
	exit(0);
  }
  else
  {
    fseek (inputFile, 0, SEEK_END);   // non-portable
    size=ftell (inputFile);
    rewind (inputFile);
    return size;
  }
}

/************************************************************************************/

static void Compress(int blocks, int warpsperblock, int dimensionality, FILE * resultFile, FILE * inPutFile, float & mediantime)
{

  hipGetLastError();  // reset error value

  // allocate CPU buffers
  //ull is unsigned long long int
  ull *cbuf = (ull *)malloc(sizeof(ull) * MAX); // uncompressed data
  if (cbuf == NULL) {
    fprintf(stderr, "cannot allocate cbuf\n"); exit(-1);
  }
  char *dbuf = (char *)malloc(sizeof(char) * ((MAX+1)/2*17)); // compressed data
  if (dbuf == NULL) {
    fprintf(stderr, "cannot allocate dbuf\n"); exit(-1);
  }
  int *cut = (int *)malloc(sizeof(int) * blocks * warpsperblock); // chunk boundaries
  if (cut == NULL) {
    fprintf(stderr, "cannot allocate cut\n"); exit(-1);
  }
  int *off = (int *)malloc(sizeof(int) * blocks * warpsperblock); // offset table
  if (off == NULL) {
    fprintf(stderr, "cannot allocate off\n"); exit(-1);
  }

  // read in trace to cbuf
  int doubles = fread(cbuf, 8, MAX, inPutFile);
  //int fileSize = getFileLength(inPutFile);
//std::cout<<"file size is "<<fileSize<<std::endl;
  //int doubles = fread(cbuf, 8, fileSize, inPutFile);

  // calculate required padding for last chunk
  int padding = ((doubles + WARPSIZE - 1) & -WARPSIZE) - doubles;
  doubles += padding;

  // determine chunk assignments per warp
  int per = (doubles + blocks * warpsperblock - 1) / (blocks * warpsperblock);
  if (per < WARPSIZE) per = WARPSIZE;
  per = (per + WARPSIZE - 1) & -WARPSIZE;
  int curr = 0, before = 0, d = 0;
  for (int i = 0; i < blocks * warpsperblock; i++) {
    curr += per;
    cut[i] = min(curr, doubles);
    if (cut[i] - before > 0) {
      d = cut[i] - before;
    }
    before = cut[i];
  }

  // set the pad values to ensure correct prediction
  if (d <= WARPSIZE) {
    for (int i = doubles - padding; i < doubles; i++) {
      cbuf[i] = 0;
    }
  } else {
    for (int i = doubles - padding; i < doubles; i++) {
      cbuf[i] = cbuf[(i & -WARPSIZE) - (dimensionality - i % dimensionality)];
    }
  }

  // allocate GPU buffers
  ull *cbufl; // uncompressed data
  char *dbufl; // compressed data
  int *cutl; // chunk boundaries
  int *offl; // offset table
  if (hipSuccess != hipMalloc((void **)&cbufl, sizeof(ull) * doubles))
    fprintf(stderr, "could not allocate cbufd\n");
  CudaTest("couldn't allocate cbufd");
  if (hipSuccess != hipMalloc((void **)&dbufl, sizeof(char) * ((doubles+1)/2*17)))
    fprintf(stderr, "could not allocate dbufd\n");
  CudaTest("couldn't allocate dbufd");
  if (hipSuccess != hipMalloc((void **)&cutl, sizeof(int) * blocks * warpsperblock))
    fprintf(stderr, "could not allocate cutd\n");
  CudaTest("couldn't allocate cutd");
  if (hipSuccess != hipMalloc((void **)&offl, sizeof(int) * blocks * warpsperblock))
    fprintf(stderr, "could not allocate offd\n");
  CudaTest("couldn't allocate offd");

  // copy buffer starting addresses (pointers) and values to constant memory
  if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(dimensionalityd), &dimensionality, sizeof(int)))
    fprintf(stderr, "copying of dimensionality to device failed\n");
  CudaTest("dimensionality copy to device failed");
  if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(cbufd), &cbufl, sizeof(void *)))
    fprintf(stderr, "copying of cbufl to device failed\n");
  CudaTest("cbufl copy to device failed");
  if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(dbufd), &dbufl, sizeof(void *)))
    fprintf(stderr, "copying of dbufl to device failed\n");
  CudaTest("dbufl copy to device failed");
  if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(cutd), &cutl, sizeof(void *)))
    fprintf(stderr, "copying of cutl to device failed\n");
  CudaTest("cutl copy to device failed");
  if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(offd), &offl, sizeof(void *)))
    fprintf(stderr, "copying of offl to device failed\n");
  CudaTest("offl copy to device failed");

  // copy CPU buffer contents to GPU
  if (hipSuccess != hipMemcpy(cbufl, cbuf, sizeof(ull) * doubles, hipMemcpyHostToDevice))
    fprintf(stderr, "copying of cbuf to device failed\n");
  CudaTest("cbuf copy to device failed");
  if (hipSuccess != hipMemcpy(cutl, cut, sizeof(int) * blocks * warpsperblock, hipMemcpyHostToDevice))
    fprintf(stderr, "copying of cut to device failed\n");
  CudaTest("cut copy to device failed");


  //record time
  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  hipEventRecord( start, 0 );

  CompressionKernel<<<blocks, WARPSIZE*warpsperblock>>>();
  CudaTest("compression kernel launch failed");

  hipEventRecord( end, 0 );
  hipEventSynchronize( end );

  float elapsedTime;
  hipEventElapsedTime( &elapsedTime, start, end );

  std::cout<<"compression time: "<<elapsedTime<<" millisecond"<<std::endl;
  mediantime=elapsedTime + mediantime;

  // transfer offsets back to CPU
  if(hipSuccess != hipMemcpy(off, offl, sizeof(int) * blocks * warpsperblock, hipMemcpyDeviceToHost))
    fprintf(stderr, "copying of off from device failed\n");
  CudaTest("off copy from device failed");

  // output header
  int num;
  int doublecnt = doubles-padding;
std::cout<<"compress block: "<<blocks<<std::endl;  
////////////////
  num = fwrite(&blocks, 4, 1, resultFile);
  assert(1 == num);
  num = fwrite(&warpsperblock, 1, 1, resultFile);
  assert(1 == num);
  num = fwrite(&dimensionality, 1, 1, resultFile);
  assert(1 == num);
  num = fwrite(&doublecnt, 4, 1, resultFile);
  assert(1 == num);
  // output offset table
  for(int i = 0; i < blocks * warpsperblock; i++) {
    int start = 0;
    if(i > 0) start = cut[i-1];
    off[i] -= ((start+1)/2*17);
    num = fwrite(&off[i], 4, 1, resultFile); // chunk's compressed size in bytes
    assert(1 == num);
  }
  // output compressed data by chunk
  for(int i = 0; i < blocks * warpsperblock; i++) {
    int offset, start = 0;
    if(i > 0) start = cut[i-1];
    offset = ((start+1)/2*17);
    // transfer compressed data back to CPU by chunk
    if (hipSuccess != hipMemcpy(dbuf + offset, dbufl + offset, sizeof(char) * off[i], hipMemcpyDeviceToHost))
      fprintf(stderr, "copying of dbuf from device failed\n");
    CudaTest("dbuf copy from device failed");
    num = fwrite(&dbuf[offset], 1, off[i], resultFile);
    assert(off[i] == num);
  }

  free(cbuf);
  free(dbuf);
  free(cut);
  free(off);

  // Cleanup in the event of success.
  hipEventDestroy( start );
  hipEventDestroy( end );

  if (hipSuccess != hipFree(cbufl))
    fprintf(stderr, "could not deallocate cbufd\n");
  CudaTest("couldn't deallocate cbufd");
  if (hipSuccess != hipFree(dbufl))
    fprintf(stderr, "could not deallocate dbufd\n");
  CudaTest("couldn't deallocate dbufd");
  if (hipSuccess != hipFree(cutl))
    fprintf(stderr, "could not deallocate cutd\n");
  CudaTest("couldn't deallocate cutd");
  if (hipSuccess != hipFree(offl))
    fprintf(stderr, "could not deallocate offd\n");
  CudaTest("couldn't deallocate offd");
}

/************************************************************************************/

static void Decompress(int blocks, int warpsperblock, int dimensionality, int doubles, FILE * resultFile, FILE * inPutFile, float & mediantimeD)
{
  hipGetLastError();  // reset error value

  // allocate CPU buffers

  char *dbuf = (char *)malloc(sizeof(char) * ((MAX+1)/2*17)); // compressed data, divided by chunk
  if (dbuf == NULL) { 
    fprintf(stderr, "cannot allocate dbuf\n"); exit(-1); 
  }
  ull *fbuf = (ull *)malloc(sizeof(ull) * MAX); // decompressed data
  if (fbuf == NULL) { 
    fprintf(stderr, "cannot allocate fbuf\n"); exit(-1);
  }
  int *cut = (int *)malloc(sizeof(int) * blocks * warpsperblock); // chunk boundaries
  if (cut == NULL) { 
    fprintf(stderr, "cannot allocate cut\n"); exit(-1);
  }
  int *off = (int *)malloc(sizeof(int) * blocks * warpsperblock); // offset table
  if(off == NULL) {
    fprintf(stderr, "cannot allocate off\n"); exit(-1);
  }

  // read in offset table
  for(int i = 0; i < blocks * warpsperblock; i++) {
    int num = fread(&off[i], 4, 1, inPutFile);
    assert(1 == num);
  }

  // calculate required padding for last chunk
  int padding = ((doubles + WARPSIZE - 1) & -WARPSIZE) - doubles;
  doubles += padding;

  // determine chunk assignments per warp
  int per = (doubles + blocks * warpsperblock - 1) / (blocks * warpsperblock); 
  if (per < WARPSIZE) per = WARPSIZE;
  per = (per + WARPSIZE - 1) & -WARPSIZE;
  int curr = 0;
  for (int i = 0; i < blocks * warpsperblock; i++) {
    curr += per;
    cut[i] = min(curr, doubles);
  }

  // allocate GPU buffers
  char *dbufl; // compressed data
  ull *fbufl; // uncompressed data
  int *cutl; // chunk boundaries
  if (hipSuccess != hipMalloc((void **)&dbufl, sizeof(char) * ((doubles+1)/2*17)))
    fprintf(stderr, "could not allocate dbufd\n");
  CudaTest("couldn't allocate dbufd");
  if (hipSuccess != hipMalloc((void **)&fbufl, sizeof(ull) * doubles))
    fprintf(stderr, "could not allocate fbufd\n");
  CudaTest("couldn't allocate fbufd");
  if (hipSuccess != hipMalloc((void **)&cutl, sizeof(int) * blocks * warpsperblock))
    fprintf(stderr, "could not allocate cutd\n");
  CudaTest("couldn't allocate cutd");

  // copy buffer starting addresses (pointers) and values to constant memory
  if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(dimensionalityd), &dimensionality, sizeof(int))) 
    fprintf(stderr, "copying of dimensionality to device failed\n");
  CudaTest("dimensionality copy to device failed");
  if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(dbufd), &dbufl, sizeof(void *)))
    fprintf(stderr, "copying of dbufl to device failed\n");
  CudaTest("dbufl copy to device failed");
  if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(fbufd), &fbufl, sizeof(void *)))
    fprintf(stderr, "copying of fbufl to device failed\n");
  CudaTest("fbufl copy to device failed");
  if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(cutd), &cutl, sizeof(void *)))
    fprintf(stderr, "copying of cutl to device failed\n");
  CudaTest("cutl copy to device failed");

  // read in input data and divide into chunks
  for(int i = 0; i < blocks * warpsperblock; i++) {
    int num, chbeg, start = 0;
    if (i > 0) start = cut[i-1];

    chbeg = ((start+1)/2*17);
    // read in this chunk of data (based on offsets)
    num = fread(&dbuf[chbeg], 1, off[i], inPutFile);
    assert(off[i] == num);
    // transfer the chunk to the GPU

    if (hipSuccess != hipMemcpy(dbufl + chbeg, dbuf + chbeg, sizeof(char) * off[i], hipMemcpyHostToDevice)) 
      fprintf(stderr, "copying of dbuf to device failed\n");
    CudaTest("dbuf copy to device failed");
  }

  // copy CPU cut buffer contents to GPU
  if (hipSuccess != hipMemcpy(cutl, cut, sizeof(int) * blocks * warpsperblock, hipMemcpyHostToDevice))
    fprintf(stderr, "copying of cut to device failed\n");
  CudaTest("cut copy to device failed");

  //record time
  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  hipEventRecord( start, 0 );

  DecompressionKernel<<<blocks, WARPSIZE*warpsperblock>>>();
  CudaTest("decompression kernel launch failed");

  hipEventRecord( end, 0 );
  hipEventSynchronize( end );

  float elapsedTime;
  hipEventElapsedTime( &elapsedTime, start, end );

 std::cout<<"decompression time: "<<elapsedTime<<" millisecond"<<std::endl;
 // timeResultFile<<elapsedTime<<"	"; 
  mediantimeD = elapsedTime + mediantimeD;   


  // transfer result back to CPU
  if (hipSuccess != hipMemcpy(fbuf, fbufl, sizeof(ull) * doubles, hipMemcpyDeviceToHost))
    fprintf(stderr, "copying of fbuf from device failed\n");
  CudaTest("fbuf copy from device failed");

  // output decompressed data
  int num = fwrite(fbuf, 8, doubles-padding, resultFile);
  assert(num == doubles-padding);

  free(dbuf);
  free(fbuf);
  free(cut);

 

  if(hipSuccess != hipFree(dbufl))
    fprintf(stderr, "could not deallocate dbufd\n");
  CudaTest("couldn't deallocate dbufd");
  if(hipSuccess != hipFree(fbufl))
    fprintf(stderr, "could not deallocate fbufl\n");
  CudaTest("couldn't deallocate fbufl");
  if(hipSuccess != hipFree(cutl))
    fprintf(stderr, "could not deallocate cutd\n");
  CudaTest("couldn't deallocate cutd");
}

/************************************************************************************/

static int VerifySystemParameters()
{
  assert(1 == sizeof(char));
  assert(4 == sizeof(int));
  assert(8 == sizeof(ull));
  int val = 1;
  assert(1 == *((char *)&val));

  int current_device = 0, sm_per_multiproc = 0; 
  int max_compute_perf = 0, max_perf_device = 0; 
  int device_count = 0, best_SM_arch = 0; 
  int arch_cores_sm[3] = { 1, 8, 32 }; 
  hipDeviceProp_t deviceProp; 

  hipGetDeviceCount(&device_count);
  if (device_count == 0) {
    fprintf(stderr, "There is no device supporting CUDA\n");
    exit(-1);
  }
   
  // Find the best major SM Architecture GPU device 
  for (current_device = 0; current_device < device_count; current_device++) { 
    hipGetDeviceProperties(&deviceProp, current_device);
    if (deviceProp.major > 0 && deviceProp.major < 9999) { 
      best_SM_arch = max(best_SM_arch, deviceProp.major); 
    }
  }
   
  // Find the best CUDA capable GPU device 
  for (current_device = 0; current_device < device_count; current_device++) { 
    hipGetDeviceProperties(&deviceProp, current_device); 
    if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
      sm_per_multiproc = 1;
    } 
    else if (deviceProp.major <= 2) { 
      sm_per_multiproc = arch_cores_sm[deviceProp.major]; 
    } 
    else { // Device has SM major > 2 
      sm_per_multiproc = arch_cores_sm[2]; 
    }
      
    int compute_perf = deviceProp.multiProcessorCount * 
                       sm_per_multiproc * deviceProp.clockRate; 
      
    if (compute_perf > max_compute_perf) { 
      // If we find GPU of SM major > 2, search only these 
      if (best_SM_arch > 2) { 
        // If device==best_SM_arch, choose this, or else pass 
        if (deviceProp.major == best_SM_arch) { 
          max_compute_perf = compute_perf; 
          max_perf_device = current_device; 
        } 
      } 
      else { 
        max_compute_perf = compute_perf; 
        max_perf_device = current_device; 
      } 
    } 
  } 
   
  hipGetDeviceProperties(&deviceProp, max_perf_device); 
  if ((deviceProp.major == 9999) && (deviceProp.minor == 9999)) {
    fprintf(stderr, "There is no CUDA capable  device\n");
    exit(-1);
  }
  if (deviceProp.major < 2) {
    fprintf(stderr, "Need at least compute capability 2.0\n");
    exit(-1);
  }
  if (deviceProp.warpSize != WARPSIZE) {
    fprintf(stderr, "Warp size must be %d\n", deviceProp.warpSize);
    exit(-1);
  }
  if ((WARPSIZE <= 0) || (WARPSIZE & (WARPSIZE-1) != 0)) {
    fprintf(stderr, "Warp size must be greater than zero and a power of two\n");
    exit(-1);
  }

  return max_perf_device;
}

/*
this function is to get mena of tempArray
*/
void getMedian(float* tempArray, int length)
{
  int j = 0;
  float tmp = 0;
  for(int i=0;i<length;i++){
    j = i;
    for(int k = i;k<length;k++){
      if(tempArray[j]>tempArray[k]){
        j = k;
      }
    }
    tmp = tempArray[i];
    tempArray[i] = tempArray[j];
    tempArray[j] = tmp;
  }
}

//multiGPU
/*
build file name
*/
char* buildFileName(char* inputFileName, int appendNum)
{
	std::string fileName;
	fileName.clear();
	fileName.append(inputFileName);
	fileName.append(".");
	char intBuf[10];
	sprintf(intBuf,"%d",appendNum);
	fileName.append(intBuf);

	char* resultFileName = new char[fileName.size() + 1];
	std::copy(fileName.begin(), fileName.end(), resultFileName);
	resultFileName[fileName.size()] = '\0';

	return resultFileName;
}

/*
This struct is used to import data into different threads
*/
struct DataStruct
{
	//compression part
	int deviceID;
	int blocks; 
	int warpsperblock; 
	int dimensionality;
	FILE * resultFile;
	FILE * inPutFile;
	float mediantime;
};
struct DataStructD
{
	//decompression part
	int deviceIDD;
	int blocksD;
	int warpsperblockD;
	int dimensionalityD;
	int doublesD;
	FILE * resultFileD;
	FILE * inPutFileD;
	float mediantimeD;
};
//this funcion is for thread operation
void* routine(void *pvoidData)
{
	DataStruct *data = (DataStruct*)pvoidData;
	hipSetDevice(data->deviceID);
	Compress(data->blocks, data->warpsperblock, data->dimensionality, data->resultFile, data->inPutFile, data->mediantime); 
	return 0;
}

void* routineD(void *pvoidData)
{
	DataStructD *data = (DataStructD*)pvoidData;
	hipSetDevice(data->deviceIDD);
	Decompress(data->blocksD, data->warpsperblockD, data->dimensionalityD, data->doublesD, data->resultFileD, data->inPutFileD, data->mediantimeD); 
	return 0;
}

/************************************************************************************/

int main(int argc, char *argv[])
{
  int blocks, warpsperblock, dimensionality;
  int device;

  fprintf(stderr, "GPU FP Compressor v2.1\n");

  device = VerifySystemParameters();
  hipSetDevice(device);

  hipFuncSetCacheConfig(reinterpret_cast<const void*>(CompressionKernel), hipFuncCachePreferL1);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(DecompressionKernel), hipFuncCachePreferL1);

  //output result
  //multiGPU
  char * resultFile;
  char * inPutFile;
  
  //output time results 
  std::ofstream timeResultFile;
  std::ofstream timeResultFileD;



  if((6 == argc) || (7 == argc)) 
  { /* compress */
    if(6 == argc) {
      dimensionality = 1;
    } else {
      dimensionality = atoi(argv[1]);
    }
	if(6 == argc) {
	  timeResultFile.open(argv[2]);
	  timeResultFileD.open(argv[5]);
    } else {
	  timeResultFile.open(argv[3]);
	  timeResultFileD.open(argv[6]);
    }
    //assert((0 < dimensionality) && (dimensionality <= WARPSIZE));
//set up for loop here
	int num, doubles;

	float mediantime[12];
	float mediantimeD[12];

	//multiGPU
	int numGPU;
	hipGetDeviceCount(&numGPU);
	DataStruct * tempData= new DataStruct[numGPU];
	DataStructD * tempDataD=new DataStructD[numGPU];
	CUTThread * thread = new CUTThread[numGPU];
	std::cout<<"There are "<<numGPU<<" GPU in this machine"<<std::endl;

	char** tempSeperateInputFile = new char*[numGPU];
	char** tempSeperateOutputFile = new char*[numGPU];
	FILE** tempInputFile = new FILE*[numGPU];
	FILE** tempOutputFile = new FILE*[numGPU];
	
	//1.41421 is sqrt(2), (tempLoop*1.41421)+1 is to get the roof value
//	for(double tempLoop = 1; tempLoop <=1024 ; tempLoop=(int)(tempLoop*1.41421)+1)
	for(double tempLoop = 100; tempLoop <=100 ; tempLoop=(int)(tempLoop*1.41421)+1)
    {			
		//initialize mediantime and mediantimeD
		for(int mediantimeCount = 0; mediantimeCount < 12 ; mediantimeCount++)
		{
			mediantime[mediantimeCount] = 0;
			mediantimeD[mediantimeCount] = 0;
		}
		for(int tempWLoop = 1; tempWLoop<=11 ;tempWLoop+=1)
//		for(int tempWLoop = 30; tempWLoop<=32 ;tempWLoop++)
		{

			if(6 == argc) 
			{
				//multiGPU
				resultFile = argv[1];	
				inPutFile = argv[3];

			}
			else 
			{
				//multiGPU
     			resultFile = argv[2];
				inPutFile = argv[4];
			}

			blocks = tempLoop;
			warpsperblock = MAXWARP;
		
			//multiGPU
			std::ifstream fileStream;
			fileStream.open(inPutFile, std::ios::in | std::ios::binary);
			int fileSize = getFileSize(&fileStream);
//			float chunkSize = (float)fileSize / (numGPU-1);//if we have 3 GPUs, divided by 2	
float chunkSize = (float)fileSize / (numGPU);//if we have 3 GPUs, divided by 2	
			chunkFile(inPutFile,inPutFile,chunkSize);		
			//multiGPU
			for(int tempGPULoop = 1 ; tempGPULoop <= numGPU ; tempGPULoop++)
			{
				//create right name for each file
				tempSeperateInputFile[tempGPULoop-1] = buildFileName(inPutFile, tempGPULoop);
				tempSeperateOutputFile[tempGPULoop-1] = buildFileName(resultFile, tempGPULoop);


//std::cout<<"this is "<<tempSeperateInputFile[tempGPULoop-1]<<" input file"<<std::endl;
//std::cout<<"this is "<<tempSeperateOutputFile[tempGPULoop-1]<<" output file"<<std::endl;
				//open files
				tempInputFile[tempGPULoop-1] = fopen(tempSeperateInputFile[tempGPULoop-1],"rb");

				tempOutputFile[tempGPULoop-1] = fopen(tempSeperateOutputFile[tempGPULoop-1],"wb");
			
				tempData[tempGPULoop-1].deviceID = tempGPULoop-1;
				tempData[tempGPULoop-1].blocks = blocks;
				tempData[tempGPULoop-1].warpsperblock = warpsperblock;
				tempData[tempGPULoop-1].dimensionality = dimensionality;
				tempData[tempGPULoop-1].resultFile = tempOutputFile[tempGPULoop-1];
				tempData[tempGPULoop-1].inPutFile = tempInputFile[tempGPULoop-1];
				tempData[tempGPULoop-1].mediantime = mediantime[tempWLoop];

				thread[tempGPULoop-1] = start_thread(routine, &tempData[tempGPULoop-1]);

			}
			for(int tempGPULoop = 1 ; tempGPULoop <= numGPU ; tempGPULoop++)
			{
				end_thread( thread[tempGPULoop-1] );	
	
				mediantime[tempWLoop] += tempData[tempGPULoop-1].mediantime;
std::cout<<tempData[tempGPULoop-1].mediantime<<" time"<<std::endl;
			
//std::cout<<"this is "<<tempGPULoop<<"finished"<<std::endl;
				fclose(tempInputFile[tempGPULoop-1]); 
				fclose(tempOutputFile[tempGPULoop-1]);
			}

		    //assert(0 == fread(&dummy, 1, 1, inPutFile));

			//decompress process here 

			if(5 == argc) 
			{
				inPutFile = argv[1];
				resultFile = argv[4];
			}
			else
			{
				inPutFile = argv[2];
				resultFile = argv[5];
			}

			//multiGPU
			for(int tempGPULoop = 1 ; tempGPULoop <= numGPU ; tempGPULoop++)
			{
				//create right name for each file
				tempSeperateInputFile[tempGPULoop-1] = buildFileName(inPutFile, tempGPULoop);
				tempSeperateOutputFile[tempGPULoop-1] = buildFileName(resultFile, tempGPULoop);
				//open files
				tempInputFile[tempGPULoop-1] = fopen(tempSeperateInputFile[tempGPULoop-1],"rb");
				tempOutputFile[tempGPULoop-1] = fopen(tempSeperateOutputFile[tempGPULoop-1],"wb");


				num = fread(&blocks, 4, 1, tempInputFile[tempGPULoop-1]);
				assert(1 == num);
				blocks &= 255;
				num = fread(&warpsperblock, 1, 1, tempInputFile[tempGPULoop-1]);
				assert(1 == num);
				warpsperblock &= 255;
				num = fread(&dimensionality, 1, 1, tempInputFile[tempGPULoop-1]);
				assert(1 == num);
				dimensionality &= 255;
				num = fread(&doubles, 4, 1, tempInputFile[tempGPULoop-1]);
				assert(1 == num);

				blocks = tempLoop;

				
				tempDataD[tempGPULoop-1].deviceIDD = tempGPULoop-1;
				tempDataD[tempGPULoop-1].blocksD = blocks;
				tempDataD[tempGPULoop-1].warpsperblockD = warpsperblock;
				tempDataD[tempGPULoop-1].dimensionalityD = dimensionality;
				tempDataD[tempGPULoop-1].doublesD = doubles;
				tempDataD[tempGPULoop-1].resultFileD = tempOutputFile[tempGPULoop-1];
				tempDataD[tempGPULoop-1].inPutFileD = tempInputFile[tempGPULoop-1];
				tempDataD[tempGPULoop-1].mediantimeD = mediantimeD[tempWLoop];

				thread[tempGPULoop-1] = start_thread(routineD, &tempDataD[tempGPULoop-1]);

			}
			for(int tempGPULoop = 1 ; tempGPULoop <= numGPU ; tempGPULoop++)
			{
				end_thread( thread[tempGPULoop-1] );	
	
				mediantimeD[tempWLoop] += tempDataD[tempGPULoop-1].mediantimeD;
			
//std::cout<<"this is "<<tempGPULoop<<"finished"<<std::endl;
				fclose(tempInputFile[tempGPULoop-1]); 
				fclose(tempOutputFile[tempGPULoop-1]);
			}

		}

		//multiGPU
		//combine all the result files into one file
		//this is for compressed file		
		if(6 == argc) 
		{
			//multiGPU
			joinFile(argv[1], argv[1]);
		}
		else 
		{
			//multiGPU
			joinFile(argv[2], argv[2]);
		}
		//this is for decompressed file
		if(5 == argc) 
		{	
			joinFile(argv[4], argv[4]);
		}
		else
		{
			joinFile(argv[5], argv[5]);
		}

		//this is the median, in fact just sort the array
		getMedian(mediantime, 11);
for(int i=1; i<12; i++)
	std::cout<<mediantime[i]<<std::endl;
	std::cout<<"the median is "<<mediantime[6]<<std::endl;
		timeResultFile<<mediantime[6];
		timeResultFile<<std::endl;
		//this is the mediantime 
		getMedian(mediantimeD, 12);

		timeResultFileD<<mediantimeD[6];
		timeResultFileD<<std::endl;

    }
  delete [] thread;
  delete [] tempSeperateInputFile;
  delete [] tempSeperateOutputFile;
  delete [] tempInputFile;
  delete [] tempOutputFile;

  }

  else {
    fprintf(stderr, "usage:\n");
    fprintf(stderr, "compress: %s blocks warps/block (dimensionality) < file.in > file.gfc\n", argv[0]);
    fprintf(stderr, "decompress: %s < file.gfc > file.out\n", argv[0]);
  }

//  fclose(resultFile); 
  return 0;
}

